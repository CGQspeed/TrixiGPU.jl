// This file is for internal test purposes only and not part of the Trixi GPU framework
// Implements launch configuration and GPU kernels via CUDA and C++
// Focus on PDE solver with DG method for 1D problems

// Include libraries and header files
#include <hip/hip_runtime.h>
#include <iostream>

// Using namespaces
using namespace std;

// Kernel configurators 
//----------------------------------------------

// CUDA kernel configurator for 1D array computing
pair<dim3, dim3> configurator_1d(void* kernelFun, int arrayLength) {
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(kernelFun));
    
    int threads = min(arrayLength, attributes.maxThreadsPerBlock);
    int blocks = ceil(static_cast<float>(arrayLength) / threads);
    
    return {dim3(blocks), dim3(threads)};
}

// CUDA kernel configurator for 2D array computing
pair<dim3, dim3> configurator_2d(void* kernelFun, int arrayWidth, int arrayHeight) {
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(kernelFun));

    int threadsPerDimension = static_cast<int>(floor(sqrt(min(arrayWidth * arrayHeight, attributes.maxThreadsPerBlock))));

    dim3 threads(threadsPerDimension, threadsPerDimension);
    dim3 blocks(ceil(static_cast<float>(arrayWidth) / threads.x), ceil(static_cast<float>(arrayHeight) / threads.y));

    return {threads, blocks};
}

// CUDA kernel configurator for 3D array computing
pair<dim3, dim3> configurator_3d(void* kernelFun, int arrayWidth, int arrayHeight, int arrayDepth) {
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(kernelFun));

    int threadsPerDimension = static_cast<int>(floor(cbrt(min(arrayWidth * arrayHeight * arrayDepth, attributes.maxThreadsPerBlock))));

    dim3 threads(threadsPerDimension, threadsPerDimension, threadsPerDimension);
    dim3 blocks(ceil(static_cast<float>(arrayWidth) / threads.x), ceil(static_cast<float>(arrayHeight) / threads.y), ceil(static_cast<float>(arrayDepth) / threads.z));

    return {threads, blocks};
}

// CUDA kernels
//----------------------------------------------

// Copy data from host to device (from double to float)
void copy_to_gpu(float*** du_device, const double*** du_host, float*** u_device, const double*** u_host, int dimX, int dimY, int dimZ) {
    int totalElements = dimX * dimY * dimZ;

    // Allocate memory on the GPU
    hipMalloc(&du_device, totalElements * sizeof(float));
    hipMalloc(&u_device, totalElements * sizeof(float));

    // Set du_device to zeros
    hipMemset(*du_device, 0, totalElements * sizeof(float));

    // Convert double data on the CPU to float and then transfer to the GPU
    float* temp_u_float = new float[totalElements];

    int idx = 0;
    for (int i = 0; i < dimX; i++) {
        for (int j = 0; j < dimY; j++) {
            for (int k = 0; k < dimZ; k++) {
                temp_u_float[idx++] = static_cast<float>(u_host[i][j][k]);
            }
        }
    }

    hipMemcpy(*u_device, temp_u_float, totalElements * sizeof(float), hipMemcpyHostToDevice);

    delete[] temp_u_float;
}



