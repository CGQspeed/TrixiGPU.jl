#include "hip/hip_runtime.h"
// This file is for internal test purposes only and not part of the Trixi GPU
// framework Implements launch configuration and GPU kernels via CUDA and C++
// Focus on PDE solver with DG method for 1D problems

// Include libraries and header files
#include "header.h"
#include <hip/hip_runtime.h>
#include <iostream>

// Using namespaces
using namespace std;

// TODO: Define matrix structs to simplify kernel calls

// Kernel configurators
//----------------------------------------------

// CUDA kernel configurator for 1D array computing
pair<dim3, dim3> configurator_1d(void *kernelFun, int arrayLength) {
    int blockSize;
    int minGridSize;

    // Get the potential block size for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       kernelFun); // Use CUDA occupancy calculator

    int threads = blockSize;
    int blocks = ceil(static_cast<float>(arrayLength) / threads);

    return {dim3(blocks), dim3(threads)};
}

// CUDA kernel configurator for 2D array computing
pair<dim3, dim3> configurator_2d(void *kernelFun, int arrayWidth, int arrayHeight) {
    int blockSize;
    int minGridSize;

    // Get the potential block size for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       kernelFun); // Use CUDA occupancy calculator

    int threadsPerDimension = static_cast<int>(sqrt(blockSize));

    dim3 threads(threadsPerDimension, threadsPerDimension);
    dim3 blocks(ceil(static_cast<float>(arrayWidth) / threads.x),
                ceil(static_cast<float>(arrayHeight) / threads.y));

    return {blocks, threads};
}

// CUDA kernel configurator for 3D array computing
pair<dim3, dim3> configurator_3d(void *kernelFun, int arrayWidth, int arrayHeight, int arrayDepth) {
    int blockSize;
    int minGridSize;

    // Get the potential block size for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       kernelFun); // Use CUDA occupancy calculator

    int threadsPerDimension = static_cast<int>(cbrt(blockSize));

    dim3 threads(threadsPerDimension, threadsPerDimension, threadsPerDimension);
    dim3 blocks(ceil(static_cast<float>(arrayWidth) / threads.x),
                ceil(static_cast<float>(arrayHeight) / threads.y),
                ceil(static_cast<float>(arrayDepth) / threads.z));

    return {blocks, threads};
}

// CUDA kernels
//----------------------------------------------

// Copy data from host to device (from double to float)
void copy_to_gpu(float ***&du_device, double ***du_host, float ***&u_device, double ***u_host,
                 int width, int height, int depth) {

    // 3D extent for allocation
    hipExtent extent = make_hipExtent(width * sizeof(float), height, depth);

    // Allocate memory for du on the GPU and set to zero
    hipPitchedPtr devDuPitchedPtr;
    hipMalloc3D(&devDuPitchedPtr, extent);
    hipMemset3D(devDuPitchedPtr, 0, extent);

    // Allocate memory for u on the GPU
    hipPitchedPtr devUPitchedPtr;
    hipMalloc3D(&devUPitchedPtr, extent);

    // Convert u from double to float and copy to GPU
    hipMemcpy3DParms copyParams = {0};
    float *temp_u_float = new float[width * height * depth];

    int idx = 0;
    for (int z = 0; z < depth; z++) {
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
                temp_u_float[idx++] = static_cast<float>(u_host[z][y][x]);
            }
        }
    }

    copyParams.srcPtr =
        make_hipPitchedPtr((void *)temp_u_float, width * sizeof(float), width, height);
    copyParams.dstPtr = devUPitchedPtr;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // Assign the pointers to the device memory
    du_device = (float ***)devDuPitchedPtr.ptr;
    u_device = (float ***)devUPitchedPtr.ptr;

    delete[] temp_u_float;
}

// Copy data from device to host (from float to double)
void copy_to_cpu(float ***du_device, double ***&du_host, float ***u_device, double ***&u_host,
                 int width, int height, int depth) {

    // 3D extent for copy
    hipExtent extent = make_hipExtent(width * sizeof(float), height, depth);

    // Temporary buffer for float data from the device
    float *temp_u_float = new float[width * height * depth];
    float *temp_du_float = new float[width * height * depth];

    hipMemcpy3DParms copyParamsU = {0};
    copyParamsU.dstPtr =
        make_hipPitchedPtr((void *)temp_u_float, width * sizeof(float), width, height);
    copyParamsU.srcPtr =
        make_hipPitchedPtr((void *)u_device, width * sizeof(float), width, height);
    copyParamsU.extent = extent;
    copyParamsU.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&copyParamsU);

    hipMemcpy3DParms copyParamsDu = {0};
    copyParamsDu.dstPtr =
        make_hipPitchedPtr((void *)temp_du_float, width * sizeof(float), width, height);
    copyParamsDu.srcPtr =
        make_hipPitchedPtr((void *)du_device, width * sizeof(float), width, height);
    copyParamsDu.extent = extent;
    copyParamsDu.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&copyParamsDu);

    // Convert float data back to double and store in u_host
    int idx = 0;
    for (int z = 0; z < depth; z++) {
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
                u_host[z][y][x] = static_cast<double>(temp_u_float[idx]);
                du_host[z][y][x] = static_cast<double>(temp_du_float[idx]);
                idx++;
            }
        }
    }

    delete[] temp_u_float;
    delete[] temp_du_float;

    // Free GPU memory
    hipFree(du_device);
    hipFree(u_device);
}

// CUDA kernel for calculating fluxes along normal direction 1
__global__ void flux_kernel(float *flux_arr, float *u, int u_dim1, int u_dim2, int u_dim3,
                            AbstractEquations equations) { // TODO
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < u_dim2 && k < u_dim3) {
        float *u_node = get_nodes_vars(u, equations, j, k); // TODO

        // This assumes you have a `flux` equivalent in C++
        float *flux_node = flux(u_node, 1, equations);

        for (int ii = 0; ii < u_dim1; ii++) {
            flux_arr[ii * u_dim2 * u_dim3 + j * u_dim3 + k] = flux_node[ii];
        }

        // Make sure to deallocate any memory you dynamically allocated
        delete[] u_node;
        delete[] flux_node;
    }
}

// CUDA kernel for calculating weak form
__global__ void weak_form_kernel(float *du, float *derivative_dhat, float *flux_arr, int du_dim1,
                                 int du_dim2, int du_dim3) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < du_dim1 && j < du_dim2 && k < du_dim3) {
        for (int ii = 0; ii < du_dim2; ii++) {
            int du_idx = i * du_dim2 * du_dim3 + j * du_dim3 + k;
            int derivative_idx = j * du_dim2 + ii;
            int flux_idx = i * du_dim2 * du_dim3 + ii * du_dim3 + k;

            du[du_idx] += derivative_dhat[derivative_idx] * flux_arr[flux_idx];
        }
    }
}

// CUDA kernel for calculating volume fluxes in direction x
__global__ void volume_flux_kernel(float *volume_flux_arr, float *u, int u_dim1, int u_dim2,
                                   int u_dim3, AbstractEquations equations) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < u_dim2 * u_dim2 && k < u_dim3) {
        int j1 = j / u_dim2;
        int j2 = j % u_dim2;

        float *u_node = get_nodes_vars(u, equations, j1, k);  // TODO
        float *u_node1 = get_nodes_vars(u, equations, j2, k); // TODO

        float *volume_flux_node = volume_flux(u_node, u_node1, 1, equations); // TODO

        for (int ii = 0; ii < u_dim1; ii++) {
            volume_flux_arr[ii * u_dim2 * u_dim2 * u_dim3 + j1 * u_dim2 * u_dim3 + j2 * u_dim3 +
                            k] = volume_flux_node[ii];
        }

        // Make sure to deallocate any memory you dynamically allocated.
        delete[] u_node;
        delete[] u_node1;
        delete[] volume_flux_node;
    }
}

// CUDA kernel for calculating symmetric and nonsymmetric fluxes in direction x
__global__ void symmetric_noncons_flux_kernel(float *symmetric_flux_arr, float *noncons_flux_arr,
                                              float *u, float *derivative_split, int u_dim1,
                                              int u_dim2, int u_dim3, AbstractEquations equations) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < u_dim2 * u_dim2 && k < u_dim3) {
        int j1 = j / u_dim2;
        int j2 = j % u_dim2;

        float *u_node = get_nodes_vars(u, equations, j1, k);
        float *u_node1 = get_nodes_vars(u, equations, j2, k);

        float *symmetric_flux_node = symmetric_flux(u_node, u_node1, 1, equations);     // TODO
        float *noncons_flux_node = nonconservative_flux(u_node, u_node1, 1, equations); // TODO

        for (int ii = 0; ii < u_dim1; ii++) {
            symmetric_flux_arr[ii * u_dim2 * u_dim2 * u_dim3 + j1 * u_dim2 * u_dim3 + j2 * u_dim3 +
                               k] = symmetric_flux_node[ii];
            noncons_flux_arr[ii * u_dim2 * u_dim2 * u_dim3 + j1 * u_dim2 * u_dim3 + j2 * u_dim3 +
                             k] = noncons_flux_node[ii] * derivative_split[j1 * u_dim2 + j2];
        }

        // Deallocate dynamically allocated memory
        delete[] u_node;
        delete[] u_node1;
        delete[] symmetric_flux_node;
        delete[] noncons_flux_node;
    }
}

// CUDA kernel for calculating volume integrals
// CUDA kernel for calculating symmetric and nonsymmetric volume integrals

// Launch CUDA kernels to calculate volume integrals