#include "hip/hip_runtime.h"
/*
This file is for internal test purposes only and is not part of the Trixi GPU framework. It
implements launch configurations and GPU kernels using CUDA and C++. The focus is on solving PDEs
with the DG method for 2D problems.
*/

// Include libraries and header files
#include "header.h"
#include <iostream>

// Using namespaces
using namespace std;

// TODO: Define matrix structs to simplify CUDA kenerls and kernel calls

// Kernel configurators
//----------------------------------------------

// CUDA kernel configurator for 1D array computing
pair<dim3, dim3> configurator_1d(void *kernelFun, int arrayLength) {
    int blockSize;
    int minGridSize;

    // Get the potential block size for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       kernelFun); // Use CUDA occupancy calculator

    int threads = blockSize;
    int blocks = ceil(static_cast<float>(arrayLength) / threads);

    return {dim3(blocks), dim3(threads)};
}

// CUDA kernel configurator for 2D array computing
pair<dim3, dim3> configurator_2d(void *kernelFun, int arrayWidth, int arrayHeight) {
    int blockSize;
    int minGridSize;

    // Get the potential block size for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       kernelFun); // Use CUDA occupancy calculator

    int threadsPerDimension = static_cast<int>(sqrt(blockSize));

    dim3 threads(threadsPerDimension, threadsPerDimension);
    dim3 blocks(ceil(static_cast<float>(arrayWidth) / threads.x),
                ceil(static_cast<float>(arrayHeight) / threads.y));

    return {blocks, threads};
}

// CUDA kernel configurator for 3D array computing
pair<dim3, dim3> configurator_3d(void *kernelFun, int arrayWidth, int arrayHeight, int arrayDepth) {
    int blockSize;
    int minGridSize;

    // Get the potential block size for maximum occupancy
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                       kernelFun); // Use CUDA occupancy calculator

    int threadsPerDimension = static_cast<int>(cbrt(blockSize));

    dim3 threads(threadsPerDimension, threadsPerDimension, threadsPerDimension);
    dim3 blocks(ceil(static_cast<float>(arrayWidth) / threads.x),
                ceil(static_cast<float>(arrayHeight) / threads.y),
                ceil(static_cast<float>(arrayDepth) / threads.z));

    return {blocks, threads};
}

// CUDA kernels
//----------------------------------------------

/*
Data Storage Decision: 1D (Flattened) vs. 3D Format on GPU

1D (Flattened) Format:
- Pros:
    - Linear memory access can provide better cache coherency and memory throughput in some cases.
    - Simplifies the indexing logic in kernels, as you only deal with a single index.
    - Easier interoperability with libraries or functions that expect linear memory.

- Cons:
    - The logic to map between 3D spatial coordinates and 1D indices might be less intuitive.
    - Can lead to divergent access patterns if neighboring threads access non-contiguous memory
locations.

3D Format:
- Pros:
    - More intuitive indexing based on spatial coordinates.
    - Can lead to coalesced memory accesses if neighboring threads access neighboring spatial
coordinates.
    - Easier to visualize and debug, especially when analyzing spatial patterns.

- Cons:
    - Might be slightly more overhead in indexing calculations.
    - Some GPU functions or libraries might expect linear memory and would require conversion.


Test both formats in the context of the specific application. Measure performance, ease of
development, and other relevant metrics. After careful consideration and based on empirical data and
specific application needs, we have currently chosen to use the 1D format.
*/

// TODO: Implement a function to convert du and u into du_host and u_host as flattened 1D arrays

// Copy data from host to device (from double to float)
void copy_to_gpu(float *&du_device, double *du_host, float *&u_device, double *u_host, int width,
                 int height, int depth) {

    // Calculate total size for the 1D array
    size_t totalSize = width * height ^ 2 * depth * sizeof(float);

    // Allocate linear memory for `du` on the GPU and set to zero
    hipMalloc((void **)&du_device, totalSize);
    hipMemset(du_device, 0, totalSize);

    // Allocate linear memory for `u` on the GPU
    hipMalloc((void **)&u_device, totalSize);

    // Convert `u` from double to float and copy to GPU in 1D format
    float *temp_u_float = new float[width * height ^ 2 * depth];
    for (int i = 0; i < width * height ^ 2 * depth; i++) {
        temp_u_float[i] = static_cast<float>(u_host[i]);
    }

    // Copy the linear memory to the GPU
    hipMemcpy(u_device, temp_u_float, totalSize, hipMemcpyHostToDevice);

    delete[] temp_u_float;
}

// Copy data from device to host (from float to double)
void copy_to_cpu(float *du_device, double *&du_host, float *u_device, double *&u_host, int width,
                 int height, int depth) {

    // Calculate total size for the 1D array
    size_t totalSize = width * height ^ 2 * depth * sizeof(float);

    // Temporary buffers for float data from the device
    float *temp_u_float = new float[width * height ^ 2 * depth];
    float *temp_du_float = new float[width * height ^ 2 * depth];

    // Copy data from device (GPU) to temporary float buffers on host (CPU)
    hipMemcpy(temp_u_float, u_device, totalSize, hipMemcpyDeviceToHost);
    hipMemcpy(temp_du_float, du_device, totalSize, hipMemcpyDeviceToHost);

    // Convert float data back to double and store in 1D host arrays
    for (int idx = 0; idx < width * height ^ 2 * depth; idx++) {
        u_host[idx] = static_cast<double>(temp_u_float[idx]);
        du_host[idx] = static_cast<double>(temp_du_float[idx]);
    }

    delete[] temp_u_float;
    delete[] temp_du_float;

    // Free GPU memory
    hipFree(du_device);
    hipFree(u_device);
}

/* // Copy data from host to device (from double to float)
void copy_to_gpu(float ***&du_device, double ***du_host, float ***&u_device, double ***u_host,
                 int width, int height, int depth) {

    // 3D extent for allocation
    hipExtent extent = make_hipExtent(width * sizeof(float), height ^ 2,
                                        depth); // We treat it as a 3D array with height = height^2

    // Allocate memory for `du` on the GPU and set to zero
    hipPitchedPtr devDuPitchedPtr;
    hipMalloc3D(&devDuPitchedPtr, extent);
    hipMemset3D(devDuPitchedPtr, 0, extent);

    // Allocate memory for `u` on the GPU
    hipPitchedPtr devUPitchedPtr;
    hipMalloc3D(&devUPitchedPtr, extent);

    // Convert `u` from double to float and copy to GPU
    hipMemcpy3DParms copyParams = {0};
    float *temp_u_float = new float[width * height ^ 2 * depth];

    int idx = 0;
    for (int z = 0; z < depth; z++) {
        for (int y = 0; y < height ^ 2; y++) {
            for (int x = 0; x < width; x++) {
                temp_u_float[idx++] = static_cast<float>(u_host[z][y][x]);
            }
        }
    }

    copyParams.srcPtr =
        make_hipPitchedPtr((void *)temp_u_float, width * sizeof(float), width, height ^ 2);
    copyParams.dstPtr = devUPitchedPtr;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // Assign the pointers to the device memory
    du_device = (float ***)devDuPitchedPtr.ptr;
    u_device = (float ***)devUPitchedPtr.ptr;

    delete[] temp_u_float;
}

// Copy data from device to host (from float to double)
void copy_to_cpu(float ***du_device, double ***&du_host, float ***u_device, double ***&u_host,
                 int width, int height, int depth) {

    // 3D extent for copy
    hipExtent extent = make_hipExtent(width * sizeof(float), height ^ 2,
                                        depth); // We treat it as a 3D array with height = height^2

    // Temporary buffer for float data from the device
    float *temp_u_float = new float[width * height ^ 2 * depth];
    float *temp_du_float = new float[width * height ^ 2 * depth];

    hipMemcpy3DParms copyParamsU = {0};
    copyParamsU.dstPtr =
        make_hipPitchedPtr((void *)temp_u_float, width * sizeof(float), width, height ^ 2);
    copyParamsU.srcPtr =
        make_hipPitchedPtr((void *)u_device, width * sizeof(float), width, height ^ 2);
    copyParamsU.extent = extent;
    copyParamsU.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&copyParamsU);

    hipMemcpy3DParms copyParamsDu = {0};
    copyParamsDu.dstPtr =
        make_hipPitchedPtr((void *)temp_du_float, width * sizeof(float), width, height ^ 2);
    copyParamsDu.srcPtr =
        make_hipPitchedPtr((void *)du_device, width * sizeof(float), width, height ^ 2);
    copyParamsDu.extent = extent;
    copyParamsDu.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&copyParamsDu);

    // Convert float data back to double and store in `u_host` and `du_host`
    int idx = 0;
    for (int z = 0; z < depth; z++) {
        for (int y = 0; y < height ^ 2; y++) {
            for (int x = 0; x < width; x++) {
                u_host[z][y][x] = static_cast<double>(temp_u_float[idx]);
                du_host[z][y][x] = static_cast<double>(temp_du_float[idx]);
                idx++;
            }
        }
    }

    delete[] temp_u_float;
    delete[] temp_du_float;

    // Free GPU memory
    hipFree(du_device);
    hipFree(u_device);
} */

// CUDA kernel for calculating fluxes along normal direction 1
__global__ void flux_kernel(float *flux_arr, float *u, int u_dim1, int u_dim2, int u_dim3,
                            AbstractEquations equations) { // TODO: `AbstractEquations`

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < u_dim2 && k < u_dim3) {
        float *u_node = get_nodes_vars(u, equations, j, k); // TODO: `get_nodes_vars`

        float *flux_node = flux(u_node, 1, equations); // TODO: `flux`

        for (int ii = 0; ii < u_dim1; ii++) {
            flux_arr[ii * u_dim2 * u_dim3 + j * u_dim3 + k] = flux_node[ii];
        }

        // Make sure to deallocate any memory you dynamically allocated
        delete[] u_node;
        delete[] flux_node;
    }
}

// CUDA kernel for calculating weak form
__global__ void weak_form_kernel(float *du, float *derivative_dhat, float *flux_arr, int du_dim1,
                                 int du_dim2, int du_dim3) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < du_dim1 && j < du_dim2 && k < du_dim3) {
        for (int ii = 0; ii < du_dim2; ii++) {
            int du_idx = i * du_dim2 * du_dim3 + j * du_dim3 + k;
            int derivative_idx = j * du_dim2 + ii;
            int flux_idx = i * du_dim2 * du_dim3 + ii * du_dim3 + k;

            du[du_idx] += derivative_dhat[derivative_idx] * flux_arr[flux_idx];
        }
    }
}

// CUDA kernel for calculating volume fluxes in direction x
__global__ void volume_flux_kernel(float *volume_flux_arr, float *u, int u_dim1, int u_dim2,
                                   int u_dim3, AbstractEquations equations) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < u_dim2 * u_dim2 && k < u_dim3) {
        int j1 = j / u_dim2;
        int j2 = j % u_dim2;

        float *u_node = get_nodes_vars(u, equations, j1, k);  // TODO: `get_nodes_vars`
        float *u_node1 = get_nodes_vars(u, equations, j2, k); // TODO: `get_nodes_vars`

        float *volume_flux_node = volume_flux(u_node, u_node1, 1, equations); // TODO: `volume_flux`

        for (int ii = 0; ii < u_dim1; ii++) {
            volume_flux_arr[ii * u_dim2 * u_dim2 * u_dim3 + j1 * u_dim2 * u_dim3 + j2 * u_dim3 +
                            k] = volume_flux_node[ii];
        }

        // Make sure to deallocate any memory you dynamically allocated
        delete[] u_node;
        delete[] u_node1;
        delete[] volume_flux_node;
    }
}

// CUDA kernel for calculating symmetric and nonsymmetric fluxes in direction x
__global__ void symmetric_noncons_flux_kernel(float *symmetric_flux_arr, float *noncons_flux_arr,
                                              float *u, float *derivative_split, int u_dim1,
                                              int u_dim2, int u_dim3, AbstractEquations equations) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < u_dim2 * u_dim2 && k < u_dim3) {
        int j1 = j / u_dim2;
        int j2 = j % u_dim2;

        float *u_node = get_nodes_vars(u, equations, j1, k);
        float *u_node1 = get_nodes_vars(u, equations, j2, k);

        float *symmetric_flux_node =
            symmetric_flux(u_node, u_node1, 1, equations); // TODO: `symmetric_flux`
        float *noncons_flux_node =
            nonconservative_flux(u_node, u_node1, 1, equations); // TODO: `nonconservative_flux`

        for (int ii = 0; ii < u_dim1; ii++) {
            symmetric_flux_arr[ii * u_dim2 * u_dim2 * u_dim3 + j1 * u_dim2 * u_dim3 + j2 * u_dim3 +
                               k] = symmetric_flux_node[ii];
            noncons_flux_arr[ii * u_dim2 * u_dim2 * u_dim3 + j1 * u_dim2 * u_dim3 + j2 * u_dim3 +
                             k] = noncons_flux_node[ii] * derivative_split[j1 * u_dim2 + j2];
        }

        // Deallocate dynamically allocated memory
        delete[] u_node;
        delete[] u_node1;
        delete[] symmetric_flux_node;
        delete[] noncons_flux_node;
    }
}

// CUDA kernel for calculating volume integrals
__global__ void volume_integral_kernel(float *du, float *derivative_split, float *volume_flux_arr,
                                       int du_dim1, int du_dim2, int du_dim3) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < du_dim1 && j < du_dim2 && k < du_dim3) {

        // The size of the second axis of `du` is used in the loop iteration
        // This assumes that the second dimension of `du` and `derivative_split` are the same
        for (int ii = 0; ii < du_dim2; ++ii) {
            du[i * du_dim2 * du_dim3 + j * du_dim3 + k] +=
                derivative_split[j * du_dim2 + ii] *
                volume_flux_arr[i * du_dim2 * du_dim2 * du_dim3 + j * du_dim2 * du_dim3 +
                                ii * du_dim3 + k];
        }
    }
}

// CUDA kernel for calculating symmetric and nonsymmetric volume integrals
__global__ void volume_integral_kernel(float *du, float *derivative_split,
                                       float *symmetric_flux_arr, float *noncons_flux_arr,
                                       int du_dim1, int du_dim2, int du_dim3) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < du_dim1 && j < du_dim2 && k < du_dim3) {
        float integral_contribution = 0.0f;

        // The size of the second axis of du is used in the loop iteration
        // This assumes that the second dimension of `du` and `derivative_split` are the same
        for (int ii = 0; ii < du_dim2; ++ii) {
            du[i * du_dim2 * du_dim3 + j * du_dim3 + k] +=
                derivative_split[j * du_dim2 + ii] *
                symmetric_flux_arr[i * du_dim2 * du_dim2 * du_dim3 + j * du_dim2 * du_dim3 +
                                   ii * du_dim3 + k];

            integral_contribution += noncons_flux_arr[i * du_dim2 * du_dim2 * du_dim3 +
                                                      j * du_dim2 * du_dim3 + ii * du_dim3 + k];
        }

        du[i * du_dim2 * du_dim3 + j * du_dim3 + k] += 0.5f * integral_contribution;
    }
}

// Launch CUDA kernels to calculate volume integrals

// CUDA kernel for prolonging two interfaces in direction x
__global__ void prolong_interfaces_kernel(float *interfaces_u, float *u, int *neighbor_ids,
                                          int interfaces_u_dim2, int interfaces_u_dim3, int u_dim2,
                                          int u_dim3) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure that we don't go out of bounds
    if (j < interfaces_u_dim2 && k < interfaces_u_dim3) {
        int left_element = neighbor_ids[k];
        int right_element = neighbor_ids[interfaces_u_dim3 + k];

        // Memory access (considering flattened arrays for simplicity)
        interfaces_u[j * interfaces_u_dim3 + k] =
            u[j * u_dim2 * u_dim3 + (u_dim2 - 1) * u_dim3 + left_element - 1];
        interfaces_u[interfaces_u_dim2 * interfaces_u_dim3 + j * interfaces_u_dim3 + k] =
            u[j * u_dim2 * u_dim3 + right_element - 1];
    }
}

// Launch CUDA kernel to prolong solution to interfaces

// CUDA kernel for calculating surface fluxes
__global__ void surface_flux_kernel(float *surface_flux_arr, float *interfaces_u,
                                    int surface_flux_arr_dim2, int surface_flux_arr_dim3,
                                    AbstractEquations equations) {

    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < surface_flux_arr_dim3) {
        float *u_ll, *u_rr;
        get_surface_node_vars(interfaces_u, equations, k, u_ll,
                              u_rr); // TODO: `get_surface_node_vars`

        float *surface_flux_node = surface_flux(u_ll, u_rr, 1, equations); // TODO: `surface_flux`

        for (int jj = 0; jj < surface_flux_arr_dim2; jj++) {
            surface_flux_arr[jj * surface_flux_arr_dim3 + k] =
                surface_flux_node[jj]; // Adjusted for flattened array
        }
    }
}

// CUDA kernel for calculating surface and both nonconservative fluxes
__global__ void surface_noncons_flux_kernel(float *surface_flux_arr, float *interfaces_u,
                                            float *noncons_left_arr, float *noncons_right_arr,
                                            int surface_flux_arr_dim3,
                                            AbstractEquations equations) {

    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < surface_flux_arr_dim3) {
        float *u_ll, *u_rr;
        get_surface_node_vars(interfaces_u, equations, k, u_ll,
                              u_rr); // TODO: `get_surface_node_vars`

        float *surface_flux_node = surface_flux(u_ll, u_rr, 1, equations); // TODO: `surface_flux`
        float *noncons_left_node =
            nonconservative_flux(u_ll, u_rr, 1, equations); // TODO: `nonconservative_flux`
        float *noncons_right_node =
            nonconservative_flux(u_rr, u_ll, 1, equations); // TODO: `nonconservative_flux`

        for (int jj = 0; jj < surface_flux_arr_dim3; ++jj) {
            surface_flux_arr[jj * surface_flux_arr_dim3 + k] =
                surface_flux_node[jj]; // Adjusted based on the 1D memory layout
            noncons_left_arr[jj * surface_flux_arr_dim3 + k] =
                noncons_left_node[jj]; // Adjusted based on the 1D memory layout
            noncons_right_arr[jj * surface_flux_arr_dim3 + k] =
                noncons_right_node[jj]; // Adjusted based on the 1D memory layout
        }
    }
}

// CUDA kernel for setting interface fluxes on orientation 1
__global__ void interface_flux_kernel(float *surface_flux_values, float *surface_flux_arr,
                                      int *neighbor_ids, int surface_flux_values_dim1,
                                      int surface_flux_values_dim3, int surface_flux_arr_dim3) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int k = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i < surface_flux_values_dim1 && k < surface_flux_arr_dim3) {
        int left_id = neighbor_ids[k];
        int right_id = neighbor_ids[surface_flux_arr_dim3 + k];

        // Assuming `surface_flux_values` and `surface_flux_arr` are 3D arrays flattened to 1D
        // The indexing will depend on how the arrays are structured in memory
        surface_flux_values[i * 2 * surface_flux_values_dim3 + 1 * surface_flux_values_dim3 +
                            left_id - 1] = surface_flux_arr[i * surface_flux_arr_dim3 + k];
        surface_flux_values[i * 2 * surface_flux_values_dim3 + right_id - 1] =
            surface_flux_arr[i * surface_flux_arr_dim3 + k];
    }
}

// CUDA kernel for setting interface fluxes on orientation 1
__global__ void interface_flux_kernel(float *surface_flux_values, float *surface_flux_arr,
                                      float *noncons_left_arr, float *noncons_right_arr,
                                      int *neighbor_ids, int surface_flux_values_dim1,
                                      int surface_flux_values_dim3, int surface_flux_arr_dim3) {

    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int k = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (i < surface_flux_values_dim1 && k < surface_flux_arr_dim3) {
        int left_id = neighbor_ids[k];
        int right_id = neighbor_ids[surface_flux_arr_dim3 + k];

        // Assuming `surface_flux_values` and `surface_flux_arr` are 3D arrays flattened to 1D
        // The indexing will depend on how the arrays are structured in memory
        surface_flux_values[i * 2 * surface_flux_values_dim3 + 1 * surface_flux_values_dim3 +
                            left_id - 1] = surface_flux_arr[i * surface_flux_arr_dim3 + k] +
                                           0.5f * noncons_left_arr[i * surface_flux_arr_dim3 + k];
        surface_flux_values[i * 2 * surface_flux_values_dim3 + right_id - 1] =
            surface_flux_arr[i * surface_flux_arr_dim3 + k] +
            0.5f * noncons_right_arr[i * surface_flux_arr_dim3 + k];
    }
}

// Launch CUDA kernels to calculate interface fluxes

// CUDA kernel for prolonging two boundaries in direction x
__global__ void prolong_boundaries_kernel(float *boundaries_u, float *u, int *neighbor_ids,
                                          int *neighbor_sides, int boundaries_u_dim2,
                                          int boundaries_u_dim3, int u_dim2, int u_dim3) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    // Assuming that 3D arrays are flattened to 1D
    if (j < boundaries_u_dim2 && k < boundaries_u_dim3) {
        int element = neighbor_ids[k];
        int side = neighbor_sides[k];

        // Indexing logic depends on how arrays are laid out in memory
        int idx_boundaries_u1 = j * boundaries_u_dim3 + k;
        int idx_boundaries_u2 =
            1 * boundaries_u_dim2 * boundaries_u_dim3 + j * boundaries_u_dim3 + k;
        int idx_u1 = j * u_dim2 * u_dim3 + (u_dim2 - 1) * u_dim3 + element - 1;
        int idx_u2 = j * u_dim2 * u_dim3 + element - 1;

        boundaries_u[idx_boundaries_u1] = (side == 1) ? u[idx_u1] : 0.0f;
        boundaries_u[idx_boundaries_u2] = (side != 1) ? u[idx_u2] : 0.0f;
    }
}

// CUDA kernel for getting last and first indices
__global__ void last_first_indices_kernel(float *lasts, float *firsts,
                                          const float *n_boundaries_per_direction, int n) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        for (int ii = 0; ii <= i; ++ii) {
            lasts[i] += n_boundaries_per_direction[ii];
        }
        firsts[i] = lasts[i] - n_boundaries_per_direction[i] + 1;
    }
}

// CUDA kernel for calculating boundary fluxes on direction 1, 2
/* __global__ void boundary_flux_kernel(float *surface_flux_values, float *boundaries_u,
                                     float *node_coordinates, float t, int *boundary_arr,
                                     int *indices_arr, int *neighbor_ids, int *neighbor_sides,
                                     int *orientations, ConditionTuple boundary_conditions,
                                     AbstractEquations equations, int length_boundary_arr,
                                     int size_surface_flux_values) {

    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k < length_boundary_arr) {
        int boundary = boundary_arr[k];
        int direction = (indices_arr[0] <= boundary) + (indices_arr[1] <= boundary);

        int neighbor = neighbor_ids[boundary];
        int side = neighbor_sides[boundary];
        int orientation = orientations[boundary];

        float *u_ll, *u_rr;
        get_surface_node_vars(boundaries_u, equations, boundary, u_ll,
                              u_rr); // TODO: `get_surface_node_vars`
        float *u_inner, *x;
        u_inner = (side == 1) ? u_ll : u_rr;
        x = get_node_coords(node_coordinates, equations, boundary);

        float *boundary_flux_node = boundary_stable_helper(
            boundary_conditions, u_inner, orientation, direction, x, t, surface_flux, equations);

        for (int ii = 0; ii < size_surface_flux_values; ++ii) {
            surface_flux_values[ii * direction + neighbor] = boundary_flux_node[ii];
        }
    }
} */

// Launch CUDA kernels to calculate boundary fluxes

__global__ void surface_integral_kernel(float *du, float *factor_arr, float *surface_flux_values,
                                        int du_dim1, int du_dim2, int du_dim3) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate a linear index for a 3D array given its dimensions and indices
    auto idx = [=](int a, int b, int c) { return c + b * du_dim1 + a * du_dim1 * du_dim2; };

    if (i < du_dim1 && j < du_dim2 && k < du_dim3) {
        if (j == 0) {
            du[idx(i, j, k)] -= surface_flux_values[idx(i, 0, k)] * factor_arr[0];
        }
        if (j == du_dim2 - 1) {
            du[idx(i, j, k)] += surface_flux_values[idx(i, 1, k)] * factor_arr[1];
        }
    }
}

// Launch CUDA kernel to calculate surface integrals

// CUDA kernel for applying inverse Jacobian
__global__ void jacobian_kernel(float *du, float *inverse_jacobian, int du_dim1, int du_dim2,
                                int du_dim3) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate a linear index for a 3D array given its dimensions and indices
    auto idx = [=](int a, int b, int c) { return c + b * du_dim1 + a * du_dim1 * du_dim2; };

    if (i < du_dim1 && j < du_dim2 && k < du_dim3) {
        du[idx(i, j, k)] *= -inverse_jacobian[k];
    }
}

// Launch CUDA kernel to apply Jacobian to reference element

//
__global__ void source_terms_kernel(float *du, float *u, float *node_coordinates, float t,
                                    int du_dim1, int du_dim2, int du_dim3,
                                    AbstractEquations equations) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate a linear index for a 3D array given its dimensions and indices.
    auto idx = [=](int a, int b, int c) { return c + b * du_dim1 + a * du_dim1 * du_dim2; };

    if (j < du_dim2 && k < du_dim3) {
        float *u_local, *x_local;

        get_nodes_vars(u_local, equations, j, k);  // TODO: `get_nodes_vars`
        get_node_coords(x_local, equations, j, k); // TODO: `get_node_coords`

        float *source_terms_node;
        source_terms(u_local, x_local, t, source_terms_node);

        for (int ii = 0; ii < du_dim1; ++ii) {
            du[idx(ii, j, k)] += source_terms_node[ii];
        }
    }
}

// Launch CUDA kernel to calculate source terms

// For tests
// --------------------------------------------------

// ... [The provided functions here] ...

// This function initializes the GPU random number generator
void createRandomArrays(float *&flux_arr, float *&derivative_dhat, int width, int height,
                        int depth) {
    size_t flux_size = width * height * depth * sizeof(float);
    size_t derivative_size = width * height * sizeof(float);

    float *host_flux_arr = new float[width * height * depth];
    float *host_derivative_dhat = new float[width * height];

    // Generate random floats on host for flux_arr
    for (int i = 0; i < width * height * depth; i++) {
        host_flux_arr[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }

    // Generate random floats on host for derivative_dhat
    for (int i = 0; i < width * height; i++) {
        host_derivative_dhat[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }

    // Allocate GPU memory
    hipMalloc((void **)&flux_arr, flux_size);
    hipMalloc((void **)&derivative_dhat, derivative_size);

    // Copy random data from host to GPU
    hipMemcpy(flux_arr, host_flux_arr, flux_size, hipMemcpyHostToDevice);
    hipMemcpy(derivative_dhat, host_derivative_dhat, derivative_size, hipMemcpyHostToDevice);

    // Clean up host memory
    delete[] host_flux_arr;
    delete[] host_derivative_dhat;
}

int main() {
    // Set temporary array size for tests
    int width = 10;
    int height = 10;
    int depth = 10;
    size_t totalSize = width * height * depth;

    // Allocate and initialize host arrays
    double *u_host = new double[totalSize];
    double *du_host = new double[totalSize];
    for (int i = 0; i < totalSize; i++) {
        u_host[i] = i * 0.5; // Just some example values
        du_host[i] = 0;      // This will remain zero before the GPU computation
    }

    // Pointers for device arrays
    float *u_device = nullptr;
    float *du_device = nullptr;

    // Copy to GPU
    copy_to_gpu(du_device, du_host, u_device, u_host, width, height, depth);

    // ... [Here to run some GPU computation here on u_device and du_device] ...

    float *flux_arr, *derivative_dhat;
    createRandomArrays(flux_arr, derivative_dhat, width, height, depth);

    AbstractEquations equations; // Initialize appropriately
    auto config2d = configurator_2d((void *)flux_kernel, height, depth);
    flux_kernel<<<config2d.first, config2d.second>>>(flux_arr, u_device, width, height, depth,
                                                     equations);

    auto config3d = configurator_3d((void *)weak_form_kernel, width, height, depth);
    weak_form_kernel<<<config3d.first, config3d.second>>>(du_device, derivative_dhat, flux_arr,
                                                          width, height, depth);

    /* volume_flux_kernel<<<config3d.first, config3d.second>>>(volume_flux_arr, u_device, width,
                                                            height, depth, equations); */

    /* symmetric_noncons_flux_kernel<<<config3d.first, config3d.second>>>(
        symmetric_flux_arr, noncons_flux_arr, u, derivative_split, int u_dim1, int u_dim2,
        int u_dim3, AbstractEquations equations); */

    /* volume_integral_kernel<<<config3d.first, config3d.second>>>(du, derivative_split,
                                                                 volume_flux_arr, du_dim1,
                                                                 du_dim2, du_dim3); */

    /* volume_integral_kernel<<<config3d.first, config3d.second>>>(du, derivative_split,
                                                                 symmetric_flux_arr,
                                                                 noncons_flux_arr, du_dim1,
                                                                 du_dim2, du_dim3); */

    /* surface_flux_kernel<<<config3d.first, config3d.second>>>(surface_flux_arr, interfaces_u,
                                                                surface_flux_arr_dim2,
                                                                surface_flux_arr_dim3,
                                                                AbstractEquations equations); */

    /* surface_noncons_flux_kernel<<<config3d.first, config3d.second>>>(
        surface_flux_arr, interfaces_u, noncons_left_arr, noncons_right_arr, surface_flux_arr_dim3,
        AbstractEquations equations); */

    /* interface_flux_kernel<<<config3d.first, config3d.second>>>(surface_flux_values,
                                                                 surface_flux_arr,
                                                                 neighbor_ids,
                                                                 surface_flux_values_dim1,
                                                                 surface_flux_values_dim3,
                                                                 surface_flux_arr_dim3); */

    /* interface_flux_kernel<<<config3d.first, config3d.second>>>(surface_flux_values,
                                                                    surface_flux_arr,
                                                                    noncons_left_arr,
                                                                    noncons_right_arr,
                                                                    neighbor_ids,
                                                                    surface_flux_values_dim1,
                                                                    surface_flux_values_dim3,
                                                                    surface_flux_arr_dim3); */

    // Copy back to CPU
    copy_to_cpu(du_device, du_host, u_device, u_host, width, height, depth);

    // Print some of the results to verify (optional)
    for (int i = 0; i < 10; i++) {
        std::cout << "u_host[" << i << "] = " << u_host[i] << ", du_host[" << i
                  << "] = " << du_host[i] << std::endl;
    }

    delete[] u_host;
    delete[] du_host;

    return 0;
}
