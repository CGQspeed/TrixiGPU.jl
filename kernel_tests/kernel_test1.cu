
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel to add two arrays element-wise
__global__ void addArrays(float *a, float *b, float *result, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) {
        result[tid] = a[tid] + b[tid];
    }
}

int main() {
    int N = 1024; // Size of arrays
    size_t size = N * sizeof(float);

    float *a, *b, *result;       // Host arrays
    float *d_a, *d_b, *d_result; // Device arrays

    // Allocate memory on the host
    a = new float[N];
    b = new float[N];
    result = new float[N];

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_result, size);

    // Copy input data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, N);

    // Check kernel attributes
    hipFuncAttributes attributes;
    hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(addArrays));
    std::cout << "Max threads per block: " << attributes.maxThreadsPerBlock << std::endl;

    // Copy result from device to host
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; ++i) {
        if (result[i] != 3.0f) {
            std::cerr << "Mismatch at element " << i << ": " << result[i] << std::endl;
            break;
        }
    }

    // Clean up
    delete[] a;
    delete[] b;
    delete[] result;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}