#include "hip/hip_runtime.h"
#include "test.h"
#include <iostream>

__global__ void addPointsKernel(Point *a, Point *b, Point *c) {
    int idx = threadIdx.x;
    c[idx].x = a[idx].x + b[idx].x;
    c[idx].y = a[idx].y + b[idx].y;
}

int main() {
    const int numPoints = 1;
    Point h_a[numPoints], h_b[numPoints], h_c[numPoints];
    Point *d_a, *d_b, *d_c;

    // Initialize host data
    h_a[0].x = 1.0f;
    h_a[0].y = 2.0f;
    h_b[0].x = 3.0f;
    h_b[0].y = 4.0f;

    // Allocate device memory
    hipMalloc(&d_a, numPoints * sizeof(Point));
    hipMalloc(&d_b, numPoints * sizeof(Point));
    hipMalloc(&d_c, numPoints * sizeof(Point));

    // Copy data to device
    hipMemcpy(d_a, h_a, numPoints * sizeof(Point), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, numPoints * sizeof(Point), hipMemcpyHostToDevice);

    // Call the kernel
    addPointsKernel<<<1, numPoints>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(h_c, d_c, numPoints * sizeof(Point), hipMemcpyDeviceToHost);

    std::cout << "Result: (" << h_c[0].x << ", " << h_c[0].y << ")\n";

    // Clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
